
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <ctype.h> 
#include <sys/time.h>

__global__ void AddArraysAtDevice(int *a, int *b, int count){
	int t_id = blockIdx.x * blockDim.x + threadIdx.x;
	if(t_id < count){
		a[t_id] += b[t_id];
	}
}

int main(){
	int count = 1000;
	int *h_a = (int*)malloc(sizeof(int) * 1000);
	int *h_b = (int*)malloc(sizeof(int) * 1000);

	for(int i=0; i<count; i++){
		h_a[i] = i;
		h_b[i] = count-i;
	}

	int *d_a, *d_b;

	if(hipMalloc(&d_a, sizeof(int)*count) != hipSuccess){
		printf("Problem with memory allocation\n");
		return 0;
	}
	if(hipMalloc(&d_b, sizeof(int)*count) != hipSuccess){
		printf("Problem with memory allocation\n");
		return 0;
	}

	if(hipMemcpy(d_a, h_a, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess){
		printf("Problem with copying from host to device\n");
		return 0;
	}

	if(hipMemcpy(d_b, h_b, sizeof(int)*count, hipMemcpyHostToDevice) != hipSuccess){
		printf("Problem with copying from host to device\n");
		return 0;
	}

	AddArraysAtDevice<<<count / 256 + 1, 256>>>(d_a, d_b, count);

	if(hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost) != hipSuccess){
		printf("Problem with copying from device to host\n");
		return 0;
	}

	printf("Numbers added on GPU!!\n");

	for(int i=0; i<count; i++){
		printf("Num %d: %d\n", i+1, h_a[i]);
	}

	free(h_a);
	free(h_b);
	hipFree(d_a);
	hipFree(d_b);
	
	return 0;
}