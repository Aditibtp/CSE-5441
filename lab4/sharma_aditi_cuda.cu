#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <ctype.h> 
#include <sys/time.h>

typedef struct Grid_boxes{
  int box_id;
  int top_n;
  int bot_n;
  int left_n;
  int right_n;
  int xc;
  int yc;
  int height;
  int width;
  int *top_list;
  int *bot_list;
  int *left_list;
  int *right_list;
} Grid_box;

#define DBL_MAX 1.7976931348623158e+308 /* max value */
#define DBL_MIN 2.2250738585072014e-308 /* min positive value */

int total_boxes = 0;
Grid_box *grid_boxes;
Grid_box *d_grid_boxes;
double *dsv_c = NULL;
double *d_dsv_c = NULL;
double epsilon = 0.1;
double affect_rate = 0.1;
double cur_min_dsv = DBL_MAX;
double cur_max_dsv = DBL_MIN;

double *h_dsv_c = NULL;

int* countGflop = 0;
int* d_countGflop = 0;



// define thread hierarchy
int num_blocks = 8;
int num_th_per_blk = 1024;


int emptyline(char *line);


void printBoxes();

int emptyline(char *line){
  int len = strlen(line);
  for(int i=0; i<len; i++){
    if(!isspace(line[i])){
      return 0;
    }
  }
  return 1;
}

__device__ int imax(int a, int b){
  return a>b ? a : b;
}

__device__ int imin(int a, int b){
  return a<b ? a : b;
}

//printing all the grid boxes

void printBoxes(){
    for(int i=0; i<total_boxes; i++){
      printf("box id %d\n", grid_boxes[i].box_id);
      printf("box temprature %lf\n", h_dsv_c[i]);
      printf("box x %d\n", grid_boxes[i].xc);
      printf("box y %d\n", grid_boxes[i].yc);
      printf("box height and width %d    %d\n", grid_boxes[i].height, grid_boxes[i].width);
      
      printf("Left neighbours: ");
      for(int j=0; j<grid_boxes[i].left_n; j++){
        printf("%d ", grid_boxes[i].left_list[j]);
      }
      printf("\nRight neighbours: ");

      for(int j=0; j<grid_boxes[i].right_n; j++){
        printf("%d ", grid_boxes[i].right_list[j]);
      }
      printf("\ntop neighbours: ");

      for(int j=0; j<grid_boxes[i].top_n; j++){
        printf("%d ", grid_boxes[i].top_list[j]);
      }
      printf("\nbottom neighbours: ");

      for(int j=0; j<grid_boxes[i].bot_n; j++){
        printf("%d ", grid_boxes[i].bot_list[j]);
      }

      printf("\n*******\n");
      
  }
}

__device__ void calculateDsvForBox(int box_index, Grid_box *grid_boxes, double *old_dsv, double *dsv_c, int total_boxes, double affect_rate, double epsilon, int* countGflop){

  int cur = box_index;
  int cxc = grid_boxes[cur].xc;
  int cyc = grid_boxes[cur].yc;
  int ch = grid_boxes[cur].height;
  int cw = grid_boxes[cur].width;
  dsv_c[cur] = 0;
  int box_peri = 0;
  
  //top neighbours
  int cur_box = 0; 
  int overlap = 0;
  int ov_end = 0;
  int ov_start = 0;

  if(grid_boxes[cur].top_n > 0){
      box_peri += cw;
      for(int tn = 0; tn<grid_boxes[cur].top_n; tn++){
        cur_box = grid_boxes[cur].top_list[tn];
        ov_start = imax(grid_boxes[cur_box].xc, cxc);
        ov_end = imin(grid_boxes[cur_box].xc + grid_boxes[cur_box].width, cxc + cw);
        overlap = ov_end - ov_start;
        dsv_c[cur] += (overlap*old_dsv[cur_box]);
        countGflop[0] +=2;
    }
  }
  
  //right neighbours
  cur_box = 0; 
  overlap = 0;
  ov_end = 0;
  ov_start = 0;

  if(grid_boxes[cur].right_n > 0){
    box_peri += ch;
    for(int rn = 0; rn<grid_boxes[cur].right_n; rn++){
        cur_box = grid_boxes[cur].right_list[rn];
        ov_start = imax(grid_boxes[cur_box].yc, cyc);
        ov_end = imin(grid_boxes[cur_box].yc + grid_boxes[cur_box].height, cyc + ch);
        overlap = ov_end - ov_start;
        dsv_c[cur] += (overlap*old_dsv[cur_box]);
        countGflop[0] +=2;
    }
}
  
  //bottom neighbours
  cur_box = 0; 
  overlap = 0;
  ov_end = 0;
  ov_start = 0;

  if(grid_boxes[cur].bot_n > 0){
    box_peri += cw;
    for(int bn = 0; bn<grid_boxes[cur].bot_n; bn++){
        cur_box = grid_boxes[cur].bot_list[bn];
        ov_start = imax(grid_boxes[cur_box].xc, cxc);
        ov_end = imin(grid_boxes[cur_box].xc + grid_boxes[cur_box].width, cxc + cw);
        overlap = ov_end - ov_start;
        //printf("bottom Neighbour and temp overlap %d %d %lf\n", overlap, cur_box, grid_boxes[cur_box].temp);
        dsv_c[cur] += (overlap*old_dsv[cur_box]);
        countGflop[0] +=2;
    }
  }
  
  //left neighbours
  cur_box = 0; 
  overlap = 0;
  ov_end = 0;
  ov_start = 0;

  if(grid_boxes[cur].left_n > 0){
    box_peri += ch;
    for(int ln = 0; ln<grid_boxes[cur].left_n; ln++){
        cur_box = grid_boxes[cur].left_list[ln];
        ov_start = imax(grid_boxes[cur_box].yc, cyc);
        ov_end = imin(grid_boxes[cur_box].yc + grid_boxes[cur_box].height, cyc + ch);
        overlap = ov_end - ov_start;
        dsv_c[cur] += (overlap*old_dsv[cur_box]);
        countGflop[0] +=2;
    }
  }
  double offset = 0;

  double cur_temp = old_dsv[cur];
  if(box_peri > 0){
    double avg_dsv = dsv_c[cur]/(double)box_peri;
    offset = ((cur_temp - avg_dsv)*affect_rate);
    countGflop[0] +=3;
  }
  dsv_c[cur] = cur_temp - offset;
}

__global__ void Compute_dsv(Grid_box *grid_boxes, double *old_dsv, double *dsv_c, int total_boxes, double affect_rate, double epsilon, int* countGflop){
  
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;
  if(t_id < total_boxes){
    calculateDsvForBox(t_id, grid_boxes, old_dsv, dsv_c, total_boxes, affect_rate, epsilon, countGflop);
  }
}

int main(int argc, char *argv[]){
  
  char line[500];
  int linecounter = 0;
  char delim[] = " \t";
  
  int i=0;
  int j=0;
  //int k=0;
  
  struct timespec start, end;
  double timediff;

  if(argc != 3){
    printf("Please provide correct number of arguments in the following order: <AFFECT_RATE> <EPSILON> <Number of threads> <INPUT_FILE>\n");
    exit(0);
  }
  
  sscanf(argv[1], "%lf", &affect_rate);
  sscanf(argv[2], "%lf", &epsilon);
  
  //reading first line containng number of boxes, rows and cols
  if(fgets(line, sizeof(line), stdin)){
    i=0;
    char *ptr = strtok(line, delim);
    //int row = 0;
    //int col = 0;
    while(ptr != NULL)
    {
      //printf("%s\n", ptr);
      if(ptr && i==0){
          total_boxes = (int) strtol(ptr, (char **)NULL, 10);
      }else if(ptr && i==1){
          //row = (int) strtol(ptr, (char **)NULL, 10);
      }else if(ptr){
          //col = (int) strtol(ptr, (char **)NULL, 10);
      }
      i++;
      ptr = strtok(NULL, delim);
      
    }

  }
  
  //do all the mallocs
  grid_boxes = (Grid_box*)malloc(sizeof(Grid_box) * total_boxes);
  dsv_c = (double*)malloc(sizeof(double) * total_boxes);
  h_dsv_c = (double*)malloc(sizeof(double) * total_boxes);
  int t=0;
  
  printf("Startting to parse files\n");
  
  while (fgets(line, sizeof(line), stdin)) {
      
      if(emptyline(line)) continue;
      
      Grid_box gb;

      i=0;
      char *ptr = strtok(line, delim);
      if(linecounter == 7) linecounter = 0;
      if(linecounter == 0){
          
          gb.box_id = (int) strtol(ptr, (char **)NULL, 10);
          
      }else if(linecounter == 1){
        
        i=0;
        while(ptr != NULL)
          {
        
          if(ptr && i==0){
            gb.yc = (int) strtol(ptr, (char **)NULL, 10);
          }else if(ptr && i==1){
              gb.xc = (int) strtol(ptr, (char **)NULL, 10);
          }else if(i==2 && ptr){
              gb.height = (int) strtol(ptr, (char **)NULL, 10);
          }else if(ptr){
              gb.width = (int) strtol(ptr, (char **)NULL, 10);
          }
          i++;
          ptr = strtok(NULL, delim);
            
         }
      }else if(linecounter == 2){
       
        j=0;
        i=0;
        while(ptr != NULL)
          {
          if(ptr && i==0){
            gb.top_n = (int) strtol(ptr, (char **)NULL, 10);
            gb.top_list = (int*)malloc(sizeof(int)*(gb.top_n));
          }else if(ptr && i>=1 && j < gb.top_n){
             gb.top_list[j] = (int) strtol(ptr, (char **)NULL, 10);
             j++;
          }
          i++;
          ptr = strtok(NULL, delim);
            
         }
      }else if(linecounter == 3){
        j=0;
        i=0;
        while(ptr != NULL)
        {
          if(ptr && i==0){
            gb.bot_n = (int) strtol(ptr, (char **)NULL, 10);
            gb.bot_list =  (int*)malloc(sizeof(int)*(gb.bot_n));
          }else if(ptr && i>=1 &&j < gb.bot_n){
             gb.bot_list[j] = (int) strtol(ptr, (char **)NULL, 10);
             j++;
          }
          i++;
          ptr = strtok(NULL, delim);
            
        }
      }else if(linecounter == 4){
        j=0;
        i=0;
        while(ptr != NULL)
          {
         if(ptr && i==0){
            gb.left_n = (int) strtol(ptr, (char **)NULL, 10);
            gb.left_list = (int*)malloc(sizeof(int)*(gb.left_n));
          }else if(ptr && i>=1  &&j < gb.left_n){
             gb.left_list[j] = (int) strtol(ptr, (char **)NULL, 10);
             j++;
          }
          i++;
          ptr = strtok(NULL, delim);
            
         }
      }else if(linecounter == 5){
        j=0;
        i=0;
        while(ptr != NULL)
          {
          if(ptr && i==0){
            gb.right_n = (int) strtol(ptr, (char **)NULL, 10);
            gb.right_list = (int*)malloc(sizeof(int)*(gb.right_n));
          }else if(ptr && i>=1  &&j < gb.right_n){
             gb.right_list[j] = (int) strtol(ptr, (char **)NULL, 10);
             j++;
          }
          i++;
          ptr = strtok(NULL, delim);
            
         }
      }else if(linecounter == 6){
          sscanf(ptr, "%lf", &h_dsv_c[t]);
          cur_max_dsv = h_dsv_c[t] > cur_max_dsv ? h_dsv_c[t] : cur_max_dsv;
          cur_min_dsv = h_dsv_c[t] < cur_min_dsv ? h_dsv_c[t] : cur_min_dsv;
      }
      linecounter++;
      if(linecounter == 7){
          grid_boxes[t] = gb;
          t++;
      }
      if(t==total_boxes)break;
  }
  
  int total_iterations = 0;
  
  
  printf("Total boxes %d\n", total_boxes);

  num_blocks = (total_boxes/512 + 1);
  
  dim3 threadsPerBlock(512, 1);
  dim3 blocksPerGrid(num_blocks + 1, 1);
  
  Grid_box *h_grid_boxes = (Grid_box*)malloc(sizeof(Grid_box) * total_boxes);
  
  memcpy(h_grid_boxes, grid_boxes, total_boxes * sizeof(Grid_box));
  
  for(int cur=0; cur < total_boxes; cur++){
    //printf("cur index %d\n", cur);
    hipMalloc(&(h_grid_boxes[cur].left_list), (grid_boxes[cur].left_n)*sizeof(int));
    hipMemcpy(h_grid_boxes[cur].left_list, grid_boxes[cur].left_list,  (grid_boxes[cur].left_n)*sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&(h_grid_boxes[cur].right_list), (grid_boxes[cur].right_n)*sizeof(int));
    hipMemcpy(h_grid_boxes[cur].right_list, grid_boxes[cur].right_list,  (grid_boxes[cur].right_n)*sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&(h_grid_boxes[cur].bot_list), (grid_boxes[cur].bot_n)*sizeof(int));
    hipMemcpy(h_grid_boxes[cur].bot_list, grid_boxes[cur].bot_list,   (grid_boxes[cur].bot_n)*sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&(h_grid_boxes[cur].top_list), ( grid_boxes[cur].top_n)*sizeof(int));
    hipMemcpy(h_grid_boxes[cur].top_list, grid_boxes[cur].top_list, ( grid_boxes[cur].top_n)*sizeof(int), hipMemcpyHostToDevice);

  }

  countGflop = (int*) malloc(sizeof(int));
  countGflop[0] = 1;

  double *d_old_dsv = NULL;

  if(hipMalloc(&d_old_dsv, sizeof(double)*total_boxes) != hipSuccess){
    printf("Problem with memory allocation for old dsv\n");
    return 0;
  }

  if(hipMalloc(&d_countGflop, sizeof(int)) != hipSuccess){
    printf("Problem with memory allocation for gflop\n");
    return 0;
  }

  if(hipMalloc(&d_grid_boxes, sizeof(Grid_box)*total_boxes) != hipSuccess){
    printf("Problem with memory allocation grid boxes\n");
    return 0;
  }
  if(hipMalloc(&d_dsv_c, sizeof(double)*total_boxes) != hipSuccess){
    printf("Problem with memory allocation dsv_c\n");
    return 0;
  }
  printf("Done with mallocs and copies\n");

  if(hipMemcpy(d_countGflop, countGflop, sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
    printf("Problem with copying from host to device gflop\n");
    return 0;
  }

  if(hipMemcpy(d_grid_boxes, h_grid_boxes, sizeof(Grid_box)*total_boxes, hipMemcpyHostToDevice) != hipSuccess){
    printf("Problem with copying from host to device grid_box\n");
    return 0;
  }
  
  if(hipMemcpy(d_old_dsv, h_dsv_c, sizeof(double)*total_boxes, hipMemcpyHostToDevice) != hipSuccess){
        printf("Problem with copying from host to device old dsv\n");
        return 0;
   }

  printf("No faults till here \n");

  int gflopcounter = 0;

  //struct timeval t_start, t_end, t_diff;
  clock_gettime(CLOCK_REALTIME,& start);

  clock_t start_clock, end_clock;
  start_clock = clock();

  time_t time_t_start;
  time_t_start = time(NULL); 
  while(1){
      total_iterations++;

      Compute_dsv<<<(total_boxes/num_th_per_blk+1), num_th_per_blk>>>(d_grid_boxes, d_old_dsv, d_dsv_c,
                       total_boxes, affect_rate, epsilon, d_countGflop);
   
      hipDeviceSynchronize();
      if(hipMemcpy(dsv_c, d_dsv_c, sizeof(double)*total_boxes, hipMemcpyDeviceToHost) != hipSuccess){
        printf("Problem with copying from device to host dsv_c\n");
        return 0;
      }

      if(hipMemcpy(countGflop, d_countGflop, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess){
        printf("Problem with copying from device to host gflop\n");
        return 0;
      }
      //printf("No faults till here with synchronise\n");
      hipDeviceSynchronize();

      cur_min_dsv =  dsv_c[0];
      cur_max_dsv =  dsv_c[0];
      h_dsv_c[0] = dsv_c[0];

      for(int curx=1; curx<total_boxes; curx++){
          h_dsv_c[curx] = dsv_c[curx];
          cur_max_dsv = max(cur_max_dsv,  dsv_c[curx]);
          cur_min_dsv = min(cur_min_dsv,  dsv_c[curx]);
      }


      gflopcounter += countGflop[0];

      if(hipMemcpy(d_grid_boxes, h_grid_boxes, sizeof(Grid_box)*total_boxes, hipMemcpyHostToDevice) != hipSuccess){
        printf("Problem with copying from host to device grid_box\n");
        return 0;
      }

      if(hipMemcpy(d_old_dsv, h_dsv_c, sizeof(double)*total_boxes, hipMemcpyHostToDevice) != hipSuccess){
        printf("Problem with copying from host to device old dsv\n");
        return 0;
      }

      countGflop[0] = 0;

      if(hipMemcpy(d_countGflop, countGflop, sizeof(int), hipMemcpyHostToDevice) != hipSuccess){
        printf("Problem with copying from host to device grid_box\n");
        return 0;
      }
      //printf("No faults till here all copying finished\n");

      int diff = (cur_max_dsv - cur_min_dsv) <= (epsilon*cur_max_dsv) ? 1 : 0;
      if(diff==1){
        break;
      }

  }

  time_t time_t_end;
  time_t_end = time(NULL); 

  //double elapsed=0;
  clock_gettime(CLOCK_REALTIME,&end);
  end_clock = (double)((clock() - start_clock));
  
  timediff = (double)((end.tv_sec - start.tv_sec)*CLOCKS_PER_SEC + ((end.tv_nsec -start.tv_nsec)/1000000));

  printf("\n********************************************************************************\n");
  printf("Total number of blocks created %d,\n", total_boxes/num_th_per_blk+1);
  printf("Total number of threads per block %d,\n", num_th_per_blk);
  printf("dissipation converged in %d iterations,\n", total_iterations);
  printf("\twith max DSV = %lf and min DSV = %lf\n", cur_max_dsv, cur_min_dsv);

  printf("\taffect rate = %lf; epsilon = %lf\n\n", affect_rate, epsilon);
  printf("elapsed convergence loop time (clock_gettime()): %lf\n", timediff);
  printf("elapsed convergence loop time (clock): %ld\n", end_clock);
  printf("elapsed convergence loop time (time_t): %ld\n", (time_t_end - time_t_start));
  printf("Total Gflop operations:   %d\n", gflopcounter);
  if((time_t_end - time_t_start) > 0){
    printf("Gflop/sec operations:   %d\n", (gflopcounter/(time_t_end - time_t_start)));
  }
  printf("\n********************************************************************************\n");

  //freeing up memory
  for(int i=0; i<total_boxes; i++){
      
      free(grid_boxes[i].left_list);
      
      free(grid_boxes[i].right_list);

      free(grid_boxes[i].top_list);

      free(grid_boxes[i].bot_list);
      
  }

  for(int cur=0; cur < total_boxes; cur++){
    hipFree(h_grid_boxes[cur].left_list);
    hipFree(h_grid_boxes[cur].right_list);
    hipFree(h_grid_boxes[cur].bot_list);
    hipFree(h_grid_boxes[cur].top_list);
  }
  free(grid_boxes);
  free(dsv_c);
  free(countGflop);
  hipFree(d_countGflop);
  hipFree(d_dsv_c);
  hipFree(d_grid_boxes);


  return 0;
}
